#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include  <string.h>
#include <stdio.h>
#include <stdlib.h> 
#define buf_size 65536
#define threads 1024

__device__ void encrypt0(char *read_buf, char *write_buf, int start_index, int end_index)
{

	for (int i = start_index; i < end_index; i++)
	{
		int m = (read_buf[i] + i) % 256;
		write_buf[i] = m;
	}
}
__device__ void decrypt0(char *read_buf, char *write_buf, int start_index, int end_index)
{
	for (int i = start_index; i < end_index; i++)
	{
		int m = (read_buf[i] - i) % 256;
		write_buf[i] = m;
	}
}
__device__ void encrypt1(char *read_buf, char *write_buf, int key, int start_index, int end_index)
{
	for (int i = start_index; i<end_index; ++i)
	{
		write_buf[i] = (read_buf[i] - key) % 256;
	}
}

__device__ void decrypt1(char *read_buf, char *write_buf, int key, int start_index, int end_index)
{
	for (int i = start_index; i<end_index; ++i)
	{
		write_buf[i] = (read_buf[i] + key) % 256;
	}
}

__device__ void encrypt2(char *read_buf, char *write_buf, int key, int start_index, int end_index)
{
	for (int i = start_index; i<end_index; ++i)
	{
		write_buf[i] = read_buf[i] ^ key;
	}
}

__device__ void decrypt2(char *read_buf, char *write_buf, int key, int start_index, int end_index)
{
	for (int i = start_index; i<end_index; ++i)
	{
		write_buf[i] = read_buf[i] ^ key;
	}
}
__device__ void encrypt3(char *read_buf, char *write_buf, int key, int start_index, int end_index)
{
	for (int i = start_index; i<end_index; ++i)
	{
		write_buf[i] = (key - read_buf[i]) % 256;
	}
}

__device__ void decrypt3(char *read_buf, char *write_buf, int key, int start_index, int end_index)
{
	for (int i = start_index; i<end_index; ++i)
	{
		write_buf[i] = (key - read_buf[i]) % 256;
	}
}
__global__ void encryption_1(char *read_buf, char *write_buf, int *read_size)
{
	int rank = threadIdx.x;
	int r2 = blockIdx.x;
	int block_size = buf_size / blockDim.x;
	int start_index, end_index;
	start_index = rank * block_size + r2*gridDim.x;
	if (start_index < (*read_size))
	{
		end_index = start_index + buf_size / (gridDim.x*blockDim.x);
		end_index = (end_index <= (*read_size)) ? end_index : (*read_size);
		printf("RANK : %d, start index : %d, end_index : %d\n", rank, start_index, end_index);
		int enc = rank % 8;


		switch (enc)
		{
		case 0: encrypt0(read_buf, write_buf, start_index, end_index);
			break;
		case 1: encrypt1(read_buf, write_buf, 0xFACA, start_index, end_index);
			break;
		case 2: encrypt2(read_buf, write_buf, 0xEFFE, start_index, end_index);
			break;
		case 3: encrypt3(read_buf, write_buf, 0xBFED, start_index, end_index);
			break;
		case 4: encrypt2(read_buf, write_buf, 0xCBDB, start_index, end_index);
			break;
		case 5: encrypt3(read_buf, write_buf, 0xDADA, start_index, end_index);
			break;
		case 6: encrypt0(read_buf, write_buf, start_index, end_index);
			break;
		case 7: encrypt1(read_buf, write_buf, 0xAFFD, start_index, end_index);
			break;
		}

	}
}
__global__ void decryption_1(char *read_buf, char *write_buf, int *read_size)
{
	int rank = threadIdx.x;
	int r2 = blockIdx.x;
	int block_size = buf_size / blockDim.x;
	int start_index, end_index;
	start_index = rank * block_size + r2*gridDim.x;
	if (start_index < (*read_size))
	{
		end_index = start_index + buf_size / (gridDim.x*blockDim.x);
		end_index = (end_index <= (*read_size)) ? end_index : (*read_size);
		printf("RANK : %d, start index : %d, end_index : %d\n", rank, start_index, end_index);
		int enc = rank % 8;

		switch (enc)
		{
		case 0: decrypt0(read_buf, write_buf, start_index, end_index);
			break;
		case 1: decrypt1(read_buf, write_buf, 0xFACA, start_index, end_index);
			break;
		case 2: decrypt2(read_buf, write_buf, 0xEFFE, start_index, end_index);
			break;
		case 3: decrypt3(read_buf, write_buf, 0xBFED, start_index, end_index);
			break;
		case 4: decrypt2(read_buf, write_buf, 0xCBDB, start_index, end_index);
			break;
		case 5: decrypt3(read_buf, write_buf, 0xDADA, start_index, end_index);
			break;
		case 6: decrypt0(read_buf, write_buf, start_index, end_index);
			break;
		case 7: decrypt1(read_buf, write_buf, 0xAFFD, start_index, end_index);
			break;
		}

	}
}
int main()
{
	FILE *fp, *fpw;
	char filename[50], buf[buf_size], new_data[buf_size];
	char *d_a, *d_b;
	fprintf(stdout, "-------------------------------------------------PARACRYPT--------------------------------------------\n\n");
	int *d_size, choice, flag = 1, read_size;
	fflush(stdout);
	do
	{
		fprintf(stdout, "1. Encrypt a File\n2. Decrypt a File\n3. Exit\n");
		fflush(stdout);
		scanf("%d", &choice);
		if (choice == 3)
			exit(0);
		else if (choice == 1 || choice == 2)
			break;
		else
			printf("Invalid Option. Try Again.\n");
	} while (true);
	fprintf(stdout, "Enter File Name : ");
	fflush(stdout);
	scanf("%s", filename);
	if ((fp = fopen(filename, "rb")) == NULL)
	{
		printf("Invalid Filename.\n");
		exit(0);
	}
	//start = time_t();
	fseek(fp, 0, SEEK_END); // seek to end of file
	int fsz = ftell(fp); // get current file pointer
	fseek(fp, 0, SEEK_SET);
	printf("\nFile Size : %d bytes\n", fsz);
	int tread = 0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	if (choice == 1)
	{
		fprintf(stdout, "Encrypting File %s.\n\n", filename);
		char nf[50] = "e_";
		strcat(nf, filename);
		fpw = fopen(nf, "w+b");
	}
	else
	{
		char nf[50];
		strcpy(nf, filename);
		nf[0] = 'd';
		fprintf(stdout, "Decrypting File %s.\n\n", filename);
		fpw = fopen(nf, "w+b");
	}

	if (choice == 1)
	{
		while (flag)
		{

			read_size = fread(buf, sizeof(char), buf_size, fp);
			if (read_size == 0)
			{
				double q = (double)tread / fsz;
				fprintf(stdout, "\rPlease Wait    ... %0.1lf%%", q * 100);
				fprintf(stdout, "\nFile Successfully Encrypted\n");
				//fprintf(stdout, "Read Size : %d\n", read_size);
				fflush(stdout);
				flag = 0;
				break;
			}
			else
			{
				hipMalloc((void**)&d_a, read_size * sizeof(char));
				hipMalloc((void**)&d_b, read_size * sizeof(char));
				hipMalloc((void **)&d_size, sizeof(int));
				double q = (double)tread / fsz;
				fprintf(stdout, "\rPlease Wait    ... %0.1lf%%", q * 100);
				tread += read_size;
				//fprintf(stdout, "Read Size : %d\n", read_size);
				fflush(stdout);
				hipMemcpy(d_a, buf, read_size * sizeof(char), hipMemcpyHostToDevice);
				hipMemcpy(d_size, &read_size, sizeof(int), hipMemcpyHostToDevice);
				encryption_1 << <8, threads >> >(d_a, d_b, d_size);
				hipMemcpy(new_data, d_b, read_size * sizeof(char), hipMemcpyDeviceToHost);
				//printf("Buffer size %d, READ sIZe :%d\n", strlen(new_data),read_size);
				// printf("%s",new_data);
				//printf("%s", new_data);
				fwrite(new_data, sizeof(char), read_size, fpw);
				hipFree(d_a);
				hipFree(d_b);
				hipFree(d_size);
			}
		}
	}
	else
	{
		while (flag)
		{

			read_size = fread(buf, sizeof(char), buf_size, fp);
			if (read_size == 0)
			{
				double q = (double)tread / fsz;
				fprintf(stdout, "\rPlease Wait    ... %0.1lf%%", q * 100);
				fprintf(stdout, "\nFile Successfully Decrypted\n");
				//fprintf(stdout, "Read Size : %d\n", read_size);
				fflush(stdout);
				flag = 0;
				break;
			}
			else
			{
				hipMalloc((void**)&d_a, read_size * sizeof(char));
				hipMalloc((void**)&d_b, read_size * sizeof(char));
				hipMalloc((void **)&d_size, sizeof(int));
				double q = (double)tread / fsz;
				fprintf(stdout, "\rPlease Wait    ... %0.1lf%%", q * 100);
				tread += read_size;
				//fprintf(stdout, "Read Size : %d\n", read_size);
				fflush(stdout);
				hipMemcpy(d_a, buf, read_size * sizeof(char), hipMemcpyHostToDevice);
				hipMemcpy(d_size, &read_size, sizeof(int), hipMemcpyHostToDevice);
				decryption_1 << <8, threads >> >(d_a, d_b, d_size);
				hipMemcpy(new_data, d_b, read_size * sizeof(char), hipMemcpyDeviceToHost);
				//printf("Buffer size %d, READ sIZe :%d\n", strlen(new_data),read_size);
				// printf("%s",new_data);
				//printf("%s", new_data);
				fwrite(new_data, sizeof(char), read_size, fpw);
				hipFree(d_a);
				hipFree(d_b);
				hipFree(d_size);
			}
		}
	}
	// end = time_t();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Time Taken : %f", elapsedTime / 1000);
	fclose(fpw);
	fclose(fp);
	getchar();
	getchar();
	return 0;
}